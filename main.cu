
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

constexpr uint32_t threads_per_block = 64;
constexpr uint32_t blocks_per_grid = 4096;
constexpr uint64_t max_N = 9;
constexpr size_t num_dir = 4;
using stack_word_t = uint16_t;

__device__ stack_word_t get_dir(stack_word_t word, uint32_t index) {
  return (word >> (index * 2)) & 0b11;
}

__device__ void set_dir(stack_word_t& word, uint32_t index, stack_word_t dir) {
  word &= ~(0b11 << (index * 2));
  word |= dir << (index * 2);
}

__global__ void oneesan_kernel(const uint64_t N, const uint16_t* const init_bits, const int32_t* const init_row,
    const int32_t* const init_col, uint64_t* const result, const uint64_t length) {
  constexpr uint32_t stack_size = 48;
  constexpr uint32_t stack_index_shift = 3;
  constexpr uint32_t stack_index_mask = (1 << stack_index_shift) - 1;
  constexpr size_t stack_word_size = (stack_size + stack_index_mask) >> stack_index_shift;
  stack_word_t dir_stack[stack_word_size];
  uint16_t bits[max_N];
  const uint64_t offset = threadIdx.x + blockIdx.x * blockDim.x;
  const uint64_t stride = gridDim.x * blockDim.x;
  uint64_t index = offset;
  if (index >= length) {
    result[offset] = 0;
    return;
  }
  int32_t row = init_row[index];
  int32_t col = init_col[index];
  uint32_t stack_index = 0;
  for (uint64_t i = 0; i <= N; ++i) {
    bits[i] = init_bits[index * (N + 1) + i];
  }
  bool first = true;
  uint64_t count = 0;
  int32_t dr[num_dir] = {0, 1, 0, -1};
  int32_t dc[num_dir] = {1, 0, -1, 0};
  while (true) {
    if (row == N && col == N) {
      ++count;
    } else {
      const uint32_t stack_index_high = stack_index >> stack_index_shift;
      const uint32_t stack_index_low = stack_index & stack_index_mask;
      const uint32_t dir = first ? 0 : get_dir(dir_stack[stack_index_high], stack_index_low) + 1;
      if (dir < num_dir) {
        first = false;
        set_dir(dir_stack[stack_index_high], stack_index_low, dir);
        int32_t next_row = row + dr[dir];
        int32_t next_col = col + dc[dir];
        if (min(next_row, next_col) < 0 || max(next_row, next_col) > N) {
          continue;
        }
        if ((bits[next_row] >> next_col) & 1) {
          continue;
        }
        bits[next_row] |= 1 << next_col;
        ++stack_index;
        row = next_row;
        col = next_col;
        first = true;
        continue;
      }
    }
    if (stack_index == 0) {
      index += stride;
      if (index >= length) {
        result[offset] = count;
        return;
      }
      row = init_row[index];
      col = init_col[index];
      for (uint64_t i = 0; i <= N; ++i) {
        bits[i] = init_bits[index * (N + 1) + i];
      }
      first = true;
    } else {
      bits[row] ^= 1 << col;
      --stack_index;
      const uint32_t stack_index_high = stack_index >> stack_index_shift;
      const uint32_t stack_index_low = stack_index & stack_index_mask;
      const uint32_t prev_dir = get_dir(dir_stack[stack_index_high], stack_index_low);
      row -= dr[prev_dir];
      col -= dc[prev_dir];
      first = false;
    }
  }
}

size_t expand_recursive(std::vector<uint16_t>& bits, std::vector<uint16_t>& current_bits,
    std::vector<int32_t>& rows, std::vector<int32_t>& cols, int32_t row, int32_t col, int32_t N, int32_t depth) {
  using std::begin;
  using std::end;
  if (row == N && col == N) {
    return 1;
  }
  if (depth == 0) {
    bits.insert(end(bits), begin(current_bits), end(current_bits));
    rows.push_back(row);
    cols.push_back(col);
    return 0;
  }
  int32_t dr[num_dir] = {0, 1, 0, -1};
  int32_t dc[num_dir] = {1, 0, -1, 0};
  size_t finished_count = 0;
  for (size_t dir = 0; dir < num_dir; ++dir) {
    auto next_row = row + dr[dir];
    auto next_col = col + dc[dir];
    if (std::min(next_row, next_col) < 0 || std::max(next_row, next_col) > N) continue;
    if ((current_bits[next_row] >> next_col) & 1) continue;
    current_bits[next_row] |= 1 << next_col;
    finished_count += expand_recursive(bits, current_bits, rows, cols, next_row, next_col, N, depth-1);
    current_bits[next_row] ^= 1 << next_col;
  }
  return finished_count;
}

#define HANDLE_ERROR(expr)                                                                \
  do {                                                                                    \
    auto err = expr;                                                                      \
    if (err != hipSuccess) {                                                             \
      std::cerr << "At: " << #expr << ": " << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl; \
    }                                                                                     \
  } while (false)

int main(int argc, char** argv) {
  int32_t N = std::stoi(argv[1]);
  int32_t expand = std::stoi(argv[2]);
  std::vector<uint16_t> bits;
  std::vector<uint16_t> current_bits(N+1, 0);
  current_bits[0] |= 1;
  std::vector<int32_t> rows, cols;
  uint64_t result = expand_recursive(bits, current_bits, rows, cols, 0, 0, N, expand);
  std::cerr << bits.size() << " " << rows.size() << std::endl;
  size_t children_count = rows.size();
  const size_t threads_per_grid = blocks_per_grid * threads_per_block;
  std::vector<uint64_t> results(threads_per_grid);
  uint16_t* bits_dev = nullptr;
  int32_t* rows_dev = nullptr;
  int32_t* cols_dev = nullptr;
  uint64_t* results_dev = nullptr;
  HANDLE_ERROR(hipMalloc(&bits_dev, bits.size() * sizeof(uint16_t)));
  HANDLE_ERROR(hipMalloc(&rows_dev, rows.size() * sizeof(int32_t)));
  HANDLE_ERROR(hipMalloc(&cols_dev, cols.size() * sizeof(int32_t)));
  HANDLE_ERROR(hipMalloc(&results_dev, results.size() * sizeof(uint64_t)));
  HANDLE_ERROR(hipMemcpy(bits_dev, bits.data(), bits.size() * sizeof(uint16_t), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(rows_dev, rows.data(), rows.size() * sizeof(int32_t), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(cols_dev, cols.data(), cols.size() * sizeof(int32_t), hipMemcpyHostToDevice));
  oneesan_kernel<<<blocks_per_grid, threads_per_block>>>(N, bits_dev, rows_dev, cols_dev, results_dev, children_count);
  HANDLE_ERROR(hipMemcpy(results.data(), results_dev, results.size() * sizeof(uint64_t), hipMemcpyDeviceToHost));
  for (auto&& num : results) {
    result += num;
  }
  std::cout << result << std::endl;
}
